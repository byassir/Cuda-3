#include "hip/hip_runtime.h"
#include <opencv/cv.h>
#include <opencv/highgui.h>
//#include <opencv2/core/core.hpp>
//#include <opencv2/highgui/highgui.hpp>
//#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include "utils.h"
#include "timer.h"

cv::Mat imageRGBA;
cv::Mat imageGrey;

uchar4        *d_rgbaImage__;
unsigned char *d_greyImage__;

size_t numRows() { return imageRGBA.rows; }
size_t numCols() { return imageRGBA.cols; }

//return types are void since any internal error will be handled by quitting
//no point in returning error codes...
//returns a pointer to an RGBA version of the input image
//and a pointer to the single channel grey-scale output
//on both the host and device
void preProcess(uchar4 **inputImage, unsigned char **greyImage,
                uchar4 **d_rgbaImage, unsigned char **d_greyImage,
                const std::string &filename) {
  //make sure the context initializes ok
  checkCudaErrors(hipFree(0));

  cv::Mat image;
  image = cv::imread(filename.c_str(), CV_LOAD_IMAGE_COLOR);
  //cv::namedWindow( "Display Image", CV_WINDOW_AUTOSIZE );
  //cv::imshow( "Display Image", image );
  //cv::waitKey(0);

  if (image.empty()) {
    std::cerr << "Couldn't open file: " << filename << std::endl;
    exit(1);
  }

  cv::cvtColor(image, imageRGBA, CV_BGR2RGBA);

  //allocate memory for the output
  imageGrey.create(image.rows, image.cols, CV_8UC1);

  //This shouldn't ever happen given the way the images are created
  //at least based upon my limited understanding of OpenCV, but better to check
  if (!imageRGBA.isContinuous() || !imageGrey.isContinuous()) {
    std::cerr << "Images aren't continuous!! Exiting." << std::endl;
    exit(1);
  }

  *inputImage = (uchar4 *)imageRGBA.ptr<unsigned char>(0);
  *greyImage  = imageGrey.ptr<unsigned char>(0);

  const size_t numPixels = numRows() * numCols();
  //allocate memory on the device for both input and output
  checkCudaErrors(hipMalloc(d_rgbaImage, sizeof(uchar4) * numPixels));
  checkCudaErrors(hipMalloc(d_greyImage, sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMemset(*d_greyImage, 0, numPixels * sizeof(unsigned char))); //make sure no memory is left laying around

  //copy input array to the GPU
  std::cout << typeid(d_rgbaImage).name() << "\n" << typeid(inputImage).name();
  checkCudaErrors(hipMemcpy(*d_rgbaImage, *inputImage, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice));

  d_rgbaImage__ = *d_rgbaImage;
  d_greyImage__ = *d_greyImage;
}

void postProcess(const std::string& output_file) {
  const int numPixels = numRows() * numCols();
  //copy the output back to the host
  checkCudaErrors(hipMemcpy(imageGrey.ptr<unsigned char>(0), d_greyImage__, sizeof(unsigned char) * numPixels, hipMemcpyDeviceToHost));

  //output the image
  cv::namedWindow( "Display Image", CV_WINDOW_AUTOSIZE );
  cv::imshow( "Display Image", imageGrey );
  cv::waitKey(0);
  cv::imwrite(output_file.c_str(), imageGrey);

  //cleanup
  hipFree(d_rgbaImage__);
  hipFree(d_greyImage__);
}

__global__ void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols) {
	int r = blockIdx.x;  //row
	int c = threadIdx.y;  //col
	uchar4 rgba = rgbaImage[r*numCols + c];
	float channelSum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
	greyImage[r * numCols + c] = channelSum;


}


void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  const dim3 blockSize(1, numCols, 1);  //TODO
  const dim3 gridSize(numRows, 1, 1);  //TODO
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}


void referenceCalculation(const uchar4* const rgbaImage, unsigned char *const greyImage, size_t numRows, size_t numCols)
{
  for (size_t r = 0; r < numRows; ++r) {
    for (size_t c = 0; c < numCols; ++c) {
      uchar4 rgba = rgbaImage[r * numCols + c];
      float channelSum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
      greyImage[r * numCols + c] = channelSum;
    }
  }
}

//include the definitions of the above functions for this homework

int main(int argc, char **argv) {
  #include "timer.h"
  uchar4        *h_rgbaImage, *d_rgbaImage;
  unsigned char *h_greyImage, *d_greyImage;

  std::string input_file;
  std::string output_file;
  if (argc == 3) {
    input_file  = std::string(argv[1]);
    output_file = std::string(argv[2]);
  }
  else {
    std::cerr << "Usage: ./hw input_file output_file" << std::endl;
    exit(1);
  }
  //load the image and give us our input and output pointers
  preProcess(&h_rgbaImage, &h_greyImage, &d_rgbaImage, &d_greyImage, input_file);


  GpuTimer timer;
  timer.Start();
  //call the students' code
  //referenceCalculation(h_rgbaImage, h_greyImage, numRows(), numCols());
  your_rgba_to_greyscale(h_rgbaImage, d_rgbaImage, d_greyImage, numRows(), numCols());
  timer.Stop();
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  printf("\n");
  int err = printf("%f msecs.\n", timer.Elapsed());

  if (err < 0) {
    //Couldn't print! Probably the student closed stdout - bad news
    std::cerr << "Couldn't print timing information! STDOUT Closed!" << std::endl;
    exit(1);
  }

  //check results and output the grey image
  postProcess(output_file);

  return 0;
}
